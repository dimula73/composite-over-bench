#include <assert.h>

#include <hip/hip_runtime.h>

typedef unsigned char quint8;

__constant__ float uint8Max;
__constant__ float uint8MaxRec1;
__constant__ float uint8MaxRec2;


__global__ void emptyKernel(quint8 *src, quint8 *dst, quint8 *mask,
                           float opacity)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void overKernel(quint8 *src, quint8 *dst, quint8 *mask,
                           float opacity)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int pi = 4 * i;

    float src_c1 = src[pi];
    float src_c2 = src[pi + 1];
    float src_c3 = src[pi + 2];
    float src_a = src[pi + 3];

    float dst_c1 = dst[pi];
    float dst_c2 = dst[pi + 1];
    float dst_c3 = dst[pi + 2];
    float dst_a = dst[pi + 3];

    src_a *= float(mask[i]) * opacity * uint8MaxRec1;

    float new_a = dst_a + (uint8Max - dst_a) * src_a * uint8MaxRec1;

    float src_blend = src_a / new_a;

    dst_c1 += src_blend * (src_c1 - dst_c1);
    dst_c2 += src_blend * (src_c2 - dst_c2);
    dst_c3 += src_blend * (src_c3 - dst_c3);

    dst[pi] = dst_c1;
    dst[pi + 1] = dst_c2;
    dst[pi + 2] = dst_c3;
    dst[pi + 3] = new_a;
}

quint8 *d_src = 0;
quint8 *d_dst = 0;
quint8 *d_mask = 0;

void initCuda(int size)
{
    hipMalloc(&d_src, 4 * size);
    hipMalloc(&d_dst, 4 * size);
    hipMalloc(&d_mask, size);
}

void freeCuda()
{
    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_mask);
}

void compositePixelsCUDA(int size, quint8 *src, quint8 *dst, quint8 *mask, quint8 opacity)
{
    float h_uint8Max = 255.0f;
    float h_uint8MaxRec1 = 1.0f/255.0f;
    float h_uint8MaxRec2 = 1.0f/(255.0f * 255.0f);

    hipMemcpyToSymbol(HIP_SYMBOL(uint8Max), &h_uint8Max, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(uint8MaxRec1), &h_uint8MaxRec1, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(uint8MaxRec2), &h_uint8MaxRec2, sizeof(float));


    hipMemcpy(d_src, src, 4 * size, hipMemcpyHostToDevice);
    hipMemcpy(d_dst, dst, 4 * size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, size, hipMemcpyHostToDevice);

    // let size be multiple of 640
    int threadsPerBlock = 640;
    int blocksPerGrid = size / threadsPerBlock;

    assert(size % threadsPerBlock == 0);

    overKernel<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst, d_mask, float(opacity)/255.0f);

    //cudaDeviceSynchronize();

    hipMemcpy(dst, d_dst, 4 * size, hipMemcpyDeviceToHost);
}

void compositePixelsCUDADataTransfers(int size, quint8 *src, quint8 *dst, quint8 *mask, quint8 opacity)
{
    float h_uint8Max = 255.0f;
    float h_uint8MaxRec1 = 1.0f/255.0f;
    float h_uint8MaxRec2 = 1.0f/(255.0f * 255.0f);

    hipMemcpyToSymbol(HIP_SYMBOL(uint8Max), &h_uint8Max, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(uint8MaxRec1), &h_uint8MaxRec1, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(uint8MaxRec2), &h_uint8MaxRec2, sizeof(float));


    hipMemcpy(d_src, src, 4 * size, hipMemcpyHostToDevice);
    hipMemcpy(d_dst, dst, 4 * size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, size, hipMemcpyHostToDevice);

    // let size be multiple of 640
    int threadsPerBlock = 640;
    int blocksPerGrid = size / threadsPerBlock;

    assert(size % threadsPerBlock == 0);

    emptyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_dst, d_mask, float(opacity)/255.0f);

    //cudaDeviceSynchronize();

    hipMemcpy(dst, d_dst, 4 * size, hipMemcpyDeviceToHost);
}
